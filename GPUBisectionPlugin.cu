#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <emmintrin.h>
#include <sys/time.h>
#include "GPUBisectionPlugin.h"


void GPUBisectionPlugin::input(std::string file) {
   std::ifstream infile(file.c_str(), std::ios::in);
   infile >> N;
}

void GPUBisectionPlugin::run() {
  //Call GPU
  gpu(N);  
}

void GPUBisectionPlugin::output(std::string file) {
}
PluginProxy<GPUBisectionPlugin> GPUBisectionPluginProxy = PluginProxy<GPUBisectionPlugin>("GPUBisection", PluginManager::getInstance());

